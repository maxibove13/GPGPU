#include "hip/hip_runtime.h"
#include "util.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#define THREAD_PER_BLOCK 32
#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace std;

__global__ void transpose_kernel_gobalMem(float* d_img_in, float* d_img_out, int width, int height) {
    
    int pixel_x, pixel_y,threadId_original,threadId_trans; //Declaro variables
    pixel_x = blockIdx.x * blockDim.x + threadIdx.x; //Indices imgx análogo a el CPU transpose
    pixel_y = blockIdx.y * blockDim.y + threadIdx.y; //Indices imgy análogo a el CPU transpose

    threadId_original = pixel_y*width+pixel_x; //Indice de acceso a la imagen original

    threadId_trans = (pixel_x*height+pixel_y);//Indice de acceso a la transpuesta
    
    if (threadId_original <= width * height)
        d_img_out[threadId_trans] = d_img_in[threadId_original];
}

__global__ void transpose_kernel_sharedMem(float* d_img_in, float* d_img_out, int width, int height) {

    __shared__ float tile[THREAD_PER_BLOCK*THREAD_PER_BLOCK]; //Defino el arrray tile en shared memory  
    
    //PASO 1: Leo variables en la imagen original por filas y copio al tile de forma coalseced por filas
    int original_pixel_x, original_pixel_y,threadId_original,threadId_tile_row;
    
    original_pixel_x = blockIdx.x  * blockDim.x + threadIdx.x;
    original_pixel_y = blockIdx.y  * blockDim.y + threadIdx.y;
    
    threadId_original = original_pixel_y * width + original_pixel_x ;//Indice de acceso a la imagen original
    threadId_tile_row = threadIdx.y * blockDim.x + threadIdx.x      ;//El block dim.x es el ancho del tile
    
    tile[threadId_tile_row]= d_img_in[threadId_original];
    __syncthreads(); // Me aseguro que se hayan copiado todos los datos al tile sino algunos threades impertientens se pueden encontrar con datos nulos
     //    Garantizado los datos en memoria compartida

    //PASO 2: Accedo por columnas al tile y calculo ese índice. 
    int threadId_tile_col;
    threadId_tile_col = threadIdx.x * blockDim.y + threadIdx.y;//El block dim.y es el height del tile

    // PASO 3: Pego en las filas de la imagen de salida de forma coalesced
    int transpose_pixel_x,transpose_pixel_y,threadId_trans;
    transpose_pixel_x = blockIdx.y * blockDim.y + threadIdx.x ;//Se accede por columnas
    transpose_pixel_y = blockIdx.x * blockDim.x + threadIdx.y ;
    threadId_trans    = transpose_pixel_x + transpose_pixel_y * height ;
    
    if (threadId_trans <= width * height)
        d_img_out[threadId_trans] = tile[threadId_tile_col];
}

// __global__ void transpose_kernel_sharedMem_fixedConflict(float* d_input, float* d_output, int width, int height){
//     __shared__ float tile[threadPerBlock]; //Defino el arrray tile en shared memory  
//     //PASO 1: Leo variables en la imagen original y copio al tile de forma coalseced
//     int original_pixel_x, original_pixel_y,threadId_original,threadId_tile_row;
    
//     original_pixel_x = blockIdx.x  * blockDim.x + threadIdx.x;
//     original_pixel_y = blockIdx.y  * blockDim.y + threadIdx.y;
//     int posicion = threadIdx.x + threadIdx.y;
//     int bl_pixel_y = 0;
//     int bl_pixel_x = 0;
//     if ( posicion < Bl_size ){
//       bl_pixel_x = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.y;
//     }else{
//       bl_pixel_x = threadIdx.x + (threadIdx.y-1)*blockDim.x + threadIdx.y;
//     }
//     tile[bl_pixel_x]= *(d_input + in_pixel_x + in_pixel_y*width );
//     __syncthreads();
//     // Segundo paso
//     if( posicion < Bl_size){
//       bl_pixel_y = threadIdx.y + threadIdx.x*(blockDim.x +1);
//     }else{
//       bl_pixel_y = threadIdx.y + (threadIdx.x-1)*(blockDim.x +1) +1;
//     }
//      // Tercer paso
//      int out_pixel_x = threadIdx.x + blockIdx.y*blockDim.y;
//      int out_pixel_y = threadIdx.y + blockIdx.x*blockDim.x;
//     *(d_output + out_pixel_x + out_pixel_y*height ) = tile[bl_pixel_y];
  
//   }



void transpose_gpu(float * img_in, int width, int height, float * img_out, int threadPerBlockx, int threadPerBlocky) {

    float *d_img_in, *d_img_out;
    int nbx;
    int nby;
    unsigned int size_img = width * height * sizeof(float);

    width % threadPerBlockx == 0 ? nbx = width / threadPerBlockx : nbx = width / threadPerBlockx + 1;
    height % threadPerBlocky == 0 ? nby = height / threadPerBlocky : nby = height / threadPerBlocky + 1;

    // Inicializo variables para medir tiempos
    CLK_CUEVTS_INIT;
    
    // Reservar memoria en la GPU
    CLK_CUEVTS_START;
    CUDA_CHK(hipMalloc((void**)&d_img_in, size_img));
    CUDA_CHK(hipMalloc((void**)&d_img_out, size_img));
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    float t_elap_cuda_malloc = t_elap_cuda;

    // copiar imagen a la GPU
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(d_img_in, img_in, size_img, hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy(d_img_out, img_out, size_img, hipMemcpyHostToDevice));
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    float t_elap_cuda_cpyHtoD = t_elap_cuda;

    // configurar grilla y lanzar kernel
    dim3 grid(nbx,nby);
    dim3 block(threadPerBlockx,threadPerBlocky);


    // Ejecuta Kernel con globalMem
    CLK_CUEVTS_START;
    transpose_kernel_gobalMem <<< grid, block >>> (d_img_in, d_img_out, width, height);
    CLK_CUEVTS_STOP;

    // Obtengo los posibles errores en la llamada al kernel
	CUDA_CHK(hipGetLastError());

	// Obligo al Kernel a llegar al final de su ejecucion y hacer obtener los posibles errores
	CUDA_CHK(hipDeviceSynchronize());

    CLK_CUEVTS_ELAPSED;
    float t_elap_cuda_kernel_globalMem = t_elap_cuda;

    CLK_CUEVTS_START;
    transpose_kernel_sharedMem <<< grid, block >>> (d_img_in, d_img_out, width, height);
    CLK_CUEVTS_STOP;

    // Obtengo los posibles errores en la llamada al kernel
	CUDA_CHK(hipGetLastError());

	// Obligo al Kernel a llegar al final de su ejecucion y hacer obtener los posibles errores
	CUDA_CHK(hipDeviceSynchronize());

    CLK_CUEVTS_ELAPSED;
    float t_elap_cuda_kernel_sharedMem = t_elap_cuda;

    // transferir resultado a la memoria principal
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(img_out, d_img_out, size_img, hipMemcpyDeviceToHost));
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    float t_elap_cuda_cpyDtoH = t_elap_cuda;

    // liberar la memoria
    CLK_CUEVTS_START;
    hipFree(d_img_in);
    hipFree(d_img_out);
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    float t_elap_cuda_free = t_elap_cuda;

    printf("Transpose adjustment timing:\n");
    printf("type:               | cudaEvents\n");
    printf("malloc:             | %06.3f ms\n", t_elap_cuda_malloc);
    printf("cpyHtoD:            | %06.3f ms\n", t_elap_cuda_cpyHtoD);
    printf("kernel globalMem:   | %06.3f ms\n", t_elap_cuda_kernel_globalMem);
    printf("kernel sharedMem:   | %06.3f ms\n", t_elap_cuda_kernel_globalMem);
    printf("cpyDtoH:            | %06.3f ms\n", t_elap_cuda_cpyDtoH);
    printf("free:               | %06.3f ms\n", t_elap_cuda_free);
    printf("TOTAL globalMem:    | %06.3f ms\n", t_elap_cuda_malloc + t_elap_cuda_cpyHtoD + t_elap_cuda_kernel_globalMem + t_elap_cuda_cpyDtoH + t_elap_cuda_free + t_elap_cuda_malloc);
    printf("TOTAL sharedMem:    | %06.3f ms\n", t_elap_cuda_malloc + t_elap_cuda_cpyHtoD + t_elap_cuda_kernel_sharedMem + t_elap_cuda_cpyDtoH + t_elap_cuda_free + t_elap_cuda_malloc);
    printf("\n");
}